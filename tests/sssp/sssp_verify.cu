#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>

#include "../../src/graph.h"
#include "../../src/kernels/cpu/sssp_pull.h"
#include "../../src/kernels/gpu/sssp_pull.cuh"

/** Forward decl. */
bool verify(const weight_t *oracle_dist, const weight_t *dist, 
        const nid_t num_nodes);

int main(int argc, char *argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " [graph.wsg]" << std::endl;
        return EXIT_FAILURE;
    }

    // Load graph.
    CSRWGraph g = load_graph_from_file(argv[1]);
    std::cout << "Loaded graph." << std::endl;

    const nid_t source_id = 0; // Arbitrarily set the highest degree node as
                               // the source node.

    // Compute oracle distance.
    weight_t *oracle_dist = nullptr;
    sssp_pull_cpu_serial(g, source_id, &oracle_dist);
    std::cout << "Computed oracle distance." << std::endl;

    // Generate init distances.
    weight_t *init_dist = new weight_t[g.num_nodes];
    #pragma omp parallel for
    for (int i = 0; i < g.num_nodes; i++)
        init_dist[i] = INF_WEIGHT;
    init_dist[source_id] = 0.0f;

    // Check SSSP CPU kernel.
    {
        weight_t *dist = nullptr;
        sssp_pull_cpu(g, epoch_sssp_pull_cpu, init_dist, &dist);

        std::cout << "Verifying SSSP CPU kernel ..." << std::endl;
        bool success = verify(oracle_dist, dist, g.num_nodes);
        std::cout << " > Verification " << (success ? "succeeded" : "failed")
            << "!" << std::endl;

        delete[] dist;
    }

    // Check SSSP GPU naive kernel.
    {
        weight_t *dist = nullptr;
        sssp_pull_gpu(g, epoch_sssp_pull_gpu_naive, init_dist, &dist);

        std::cout << "Verifying SSSP GPU naive kernel ..." << std::endl;
        bool success = verify(oracle_dist, dist, g.num_nodes);
        std::cout << " > Verification " << (success ? "succeeded" : "failed")
            << "!" << std::endl;

        delete[] dist;
    }

    // Check SSSP GPU warp min kernel.
    {
        weight_t *dist = nullptr;
        sssp_pull_gpu(g, epoch_sssp_pull_gpu_warp_min, init_dist, &dist);

        std::cout << "Verifying SSSP GPU warp min kernel ..." << std::endl;
        bool success = verify(oracle_dist, dist, g.num_nodes);
        std::cout << " > Verification " << (success ? "succeeded" : "failed")
            << "!" << std::endl;

        delete[] dist;
    }

    // Check SSSP GPU block min kernel.
    {
        weight_t *dist = nullptr;
        sssp_pull_gpu(g, epoch_sssp_pull_gpu_block_min, init_dist, &dist);

        std::cout << "Verifying SSSP GPU block min kernel ..." << std::endl;
        bool success = verify(oracle_dist, dist, g.num_nodes);
        std::cout << " > Verification " << (success ? "succeeded" : "failed")
            << "!" << std::endl;

        delete[] dist;
    }

    return EXIT_SUCCESS;
}

/**
 * Verifies that the computed distances is the same as the oracle distance.
 * Parameters:
 *   - oracle_dist <- correct distance.
 *   - dist        <- computed distance (distance to check).
 *   - num_nodes   <- number of nodes in the graph.
 * Returns:
 *   true if the computed distance is correct.
 */
bool verify(const weight_t *oracle_dist, const weight_t *dist,
        const nid_t num_nodes
) {
    bool is_correct = true;

    for (nid_t nid = 0; nid < num_nodes; nid++) {
        if (oracle_dist[nid] != dist[nid]) {
            std::cout << " > " << nid << ": " 
                << dist[nid] << " != " << oracle_dist[nid] << std::endl;
            is_correct = false;
        }
    }

    return is_correct;
}
