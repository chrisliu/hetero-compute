#include "hip/hip_runtime.h"
/**
 * Implementations of CPU and GPU SSSP pull with data race.
 */

#include <omp.h> 


#include "gapbs.h"
#include "util.h"

// Forward decl.
void kernel_sssp_pull_cpu(const wgraph_t &g, weight_t *dist, const int tid, 
        const int num_threads, int &updated);
void sssp_pull_cpu(const wgraph_t &g, weight_t **ret_dist);
void sssp_pull_gpu(const wgraph_t &g, weight_t **ret_dist);
__global__ void sssp_pull_gpu_impl(const nid_t *index, 
    const cu_wnode_t *neighbors, const int num_nodes, weight_t *dist, 
    int *updated);

int main(int argc, char *argv[]) {
    // Obtain command line configs.
    CLBase cli(argc, argv);
    if (not cli.ParseArgs()) { return EXIT_FAILURE; }

    // Build ordered graph (by descending degree).
    WeightedBuilder b(cli);
    wgraph_t g = b.MakeGraph();
    wgraph_t ordered_g = b.RelabelByDegree(g);
    //wgraph_t ordered_g = b.MakeGraph();

    // Run SSSP.
    weight_t *distances = nullptr;
    sssp_pull_cpu(ordered_g, &distances);
    /*sssp_pull_gpu(ordered_g, &distances);*/

    if (cli.scale() <= 4) {
        std::cout << "node neighbors" << std::endl;
        for (int i = 0; i < ordered_g.num_nodes(); i++) {
            std::cout << " > node " << i << std::endl;
            for (auto &out_nei : ordered_g.out_neigh(i)) {
                std::cout << "    > node " << out_nei.v << ": " << out_nei.w
                    << std::endl;
            }
        }

        std::cout << "node: distance" << std::endl;
        for (int i = 0; i < ordered_g.num_nodes(); i++)
            std::cout << " > " << i << ": " << distances[i] << std::endl;
    }

    //WeightedWriter w(ordered_g);
    //w.WriteGraph("graph.wel");

    //WeightedReader r("graph.wel");

    return EXIT_SUCCESS;
}

/******************************************************************************
 ***** Kernels ****************************************************************
 ******************************************************************************/

/**
 * Runs SSSP kernel on CPU. Synchronization occurs in serial.
 * Parameters:
 *   - g        <- graph.
 *   - ret_dist <- pointer to the address of the return distance array.
 */
void sssp_pull_cpu(const wgraph_t &g, weight_t **ret_dist) {
    weight_t *dist = new weight_t[g.num_nodes()];

    #pragma omp parallel for
    for (int i = 0; i < g.num_nodes(); i++)
        dist[i] = MAX_WEIGHT;

    // Arbitrary: Set lowest degree node as source.
    dist[0] = 0;

    // Start kernel.
    std::cout << "Starting kernel ..." << std::endl;
    Timer timer; timer.Start();

    int updated = 1;

    while (updated != 0) {
        updated = 0;

        #pragma omp parallel
        {
            kernel_sssp_pull_cpu(g, dist, omp_get_thread_num(), 
                    omp_get_num_threads(), updated);
        }

        // Implicit OMP BARRIER here (see "implicit barrier at end of parallel 
        // region").
    }

    timer.Stop();
    std::cout << "Kernel completed in: " << timer.Millisecs() << " ms."
        << std::endl;

    // Assign output.
    *ret_dist = dist;
}

void sssp_pull_gpu(const wgraph_t &g, weight_t **ret_dist) {
    /// Setup.
    std::cout << "Setting up ..." << std::endl;
    // Copy graph.
    std::cout << " > Copying graph ..." << std::endl;
    nid_t      *index     = nullptr;
    cu_wnode_t *neighbors = nullptr;
    wgraph_to_cugraph(g, &index, &neighbors);
    size_t index_size     = g.num_nodes() * sizeof(nid_t);
    size_t neighbors_size = 2 * g.num_edges() * sizeof(cu_wnode_t);

    nid_t      *cu_index     = nullptr;
    cu_wnode_t *cu_neighbors = nullptr;
    hipMalloc((void **) &cu_index, index_size);
    hipMalloc((void **) &cu_neighbors, neighbors_size);
    hipMemcpy(cu_index, index, index_size, hipMemcpyHostToDevice);
    hipMemcpy(cu_neighbors, neighbors, neighbors_size, hipMemcpyHostToDevice);

    delete[] index; delete[] neighbors;

    // Distance and update counter.
    std::cout << " > Initializing distance and update counter ..." << std::endl;
    int *cu_updated = nullptr;
    hipMalloc((void **) &cu_updated, sizeof(int));
    
    weight_t *dist = new weight_t[g.num_nodes()];
    #pragma omp parallel for
    for (int i = 0; i < g.num_nodes(); i++)
        dist[i] = MAX_WEIGHT;
    dist[0] = 0;
    weight_t *cu_dist = nullptr;
    size_t dist_size = g.num_nodes() * sizeof(weight_t);
    hipMalloc((void **) &cu_dist, dist_size);
    hipMemcpy(cu_dist, dist, dist_size, hipMemcpyHostToDevice);

    // Actual kernel run.
    int updated = 1;

    std::cout << "Starting kernel ..." << std::endl;
    Timer timer; timer.Start();

    while (updated != 0) {
        hipMemset(cu_updated, 0, sizeof(int));

        sssp_pull_gpu_impl<<<1, 8>>>(cu_index, cu_neighbors, g.num_nodes(),
                cu_dist, cu_updated);

        hipMemcpy(&updated, cu_updated, sizeof(int), hipMemcpyDeviceToHost);
    }

    timer.Stop();
    std::cout << "Kernel completed in: " << timer.Millisecs() << " ms."
        << std::endl;

    // Copy distances.
    std::cout << "Copying output ..." << std::endl;
    hipMemcpy(dist, cu_dist, dist_size, hipMemcpyDeviceToHost);
    *ret_dist = dist;

    // Free memory.
    std::cout << "Freeing memory ..." << std::endl;
    hipFree(cu_index);
    hipFree(cu_neighbors);
    hipFree(cu_updated);
    hipFree(cu_dist);
}

/******************************************************************************
 ***** Epoch Kernels **********************************************************
 ******************************************************************************/

/**
 * Runs SSSP pull on CPU for one epoch.
 * Parameters:
 *   - g           <- graph.
 *   - dist        <- input distances and output distances computed this 
 *                    epoch.
 *   - tid         <- processor id.
 *   - num_threads <- number of processors.
 *   - updated     <- global counter of number of nodes updated.
 */
void kernel_sssp_pull_cpu(const wgraph_t &g, weight_t *dist, const int tid,
        const int num_threads, int &updated
) {
    int local_updated = 0;

    // Propagate, reduce, and apply.
    for (int nid = tid; nid < g.num_nodes(); nid += num_threads) {
        weight_t new_dist = dist[nid];

        // Find shortest candidate distance.
        for (wnode_t nei : g.in_neigh(nid)) {
            weight_t prop_dist = dist[nei.v] + nei.w;
            new_dist = std::min(prop_dist, new_dist);
        }

        // Update distance if applicable.
        if (new_dist != dist[nid]) {
            dist[nid] = new_dist;
            local_updated++;
        }
    }

    // Push update count.
    #pragma omp atomic
    updated += local_updated;
}

/**
 * Runs SSSP pull on GPU for one epoch.
 * Parameters:
 *   - index     <- graph index returned by deconstruct_wgraph().
 *   - neighbors <- graph neighbors returned by deconstruct_wgraph().
 *   - num_nodes <- number of nodes in this graph.
 *   - dist      <- input distance and output distances computed this epoch.
 *   - updated   <- global counter on number of nodes updated.
 */
__global__ 
void sssp_pull_gpu_impl(const nid_t *index, const cu_wnode_t *neighbors, 
        const int num_nodes, weight_t *dist, int *updated
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = gridDim.x * blockDim.x;

    int local_updated = 0;

    for (int nid = tid; nid < num_nodes; nid += num_threads) {
        weight_t new_dist = dist[nid];

        // Find shartest candiadte distance.
        for (int i = index[nid]; i < index[nid + 1]; i++) {
            weight_t prop_dist = dist[neighbors[i].v] + neighbors[i].w;
            new_dist = min(prop_dist, new_dist);
        }

        // Update distance if applicable.
        if (new_dist != dist[nid]) {
            dist[nid] = new_dist;
            local_updated++;
        }
    }

    // Push update count.
    atomicAdd(updated, local_updated);
}
